#include "hip/hip_runtime.h"
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hashiru_cuda.cuh"

__device__ void cuda_hash(const char *in, const int len, char *out)
{
   char c = 0;
   for(int i = 0; i < len; i++)
   {
       c += (char)in[i];
   }
   c = 97 + c % 26;
   out[0] = c;
   for(int i = 1; i < 32; i++)
   {
       out[i] = 'F';
   }
   out[32] = '\0';
}

__global__ void cudaCrackHashKernel(const char *dict, const int max_length, const int dict_size, const char *to_crack, int *correct_idx)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char *current, *cur_hash = (char *)malloc(33 * sizeof(char));
    int equal, len;
    char *c;
    while(idx < dict_size)
    {
        current = (char *) (dict + idx * (max_length + 1));
        len = 0;
        c = current;
        while(*c != '\0')
        {
            len++;
            c++;
        }
        memset(cur_hash, 0, 33);
        cuda_hash(current, len, cur_hash);
        // Super sketchy strcmp implementation.  Not parallel
        // and not efficient, but hopefully it should work.
        equal = 1;
        for(int i = 0; i < 32; i++)
        { 
            if(to_crack[i] != cur_hash[i]) equal = 0;
        }
        if(equal)
        { 
            *correct_idx = idx;
            break;
        }
        idx += blockDim.x * gridDim.x;
    }
}

void cudaCallCrackHashKernel(const unsigned int blocks,
         const unsigned int threadsPerBlock,
         const char *dict,
         const int max_length,
         const int dict_size,
         const char *to_crack,
         int *correct_idx)
{
    cudaCrackHashKernel<<<blocks, threadsPerBlock>>>(dict, max_length, dict_size, to_crack, correct_idx);
}
