#include "hip/hip_runtime.h"
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hashiru_cuda.cuh"

// A GPU version of the toy hash function from earlier.
// Could be done in a more parallel manner, but at this
// time I just want it to work.
__device__ void cuda_hash(const char *in, const int len, char *out)
{
   char c = 0;
   for(int i = 0; i < len; i++)
   {
       c += (char)in[i];
   }
   c = 97 + c % 26;
   out[0] = c;
   for(int i = 1; i < 32; i++)
   {
       out[i] = 'F';
   }
   out[32] = '\0';
}

__global__ void cudaCrackHashKernel(const char *dict, const int max_length, const int dict_size, const char *to_crack, int *correct_idx)
{
    // Calculate this thread's index.
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Set aside some memory for the GPU hash function to write
    // to.  Suboptimal, but it works.
    char *current, *cur_hash = (char *)malloc(33 * sizeof(char));
    int equal, len;
    char *c;
    // Iterate over the whole dictionary.
    while(idx < dict_size)
    {
        // Get the current word for consideration.
        current = (char *) (dict + idx * (max_length + 1));
        // Calculate its length in a loop.  Again, janky and not
        // parallel, but it works.
        len = 0;
        c = current;
        while(*c != '\0')
        {
            len++;
            c++;
        }
        // Nuke the hash buffer, and call the GPU hash function.
        memset(cur_hash, 0, 33);
        cuda_hash(current, len, cur_hash);
        // Super sketchy strcmp implementation.  Not parallel
        // and not efficient, but hopefully it should work.
        equal = 1;
        for(int i = 0; i < 32; i++)
        { 
            if(to_crack[i] != cur_hash[i]) equal = 0;
        }
        // Only if you stumble across the answer do you update
        // correct_idx.  If a collision occurs, it only matters
        // that one of the correct answers gets written, not
        // which one.
        if(equal)
        { 
            *correct_idx = idx;
            break;
        }
        idx += blockDim.x * gridDim.x;
    }
}

void cudaCallCrackHashKernel(const unsigned int blocks,
         const unsigned int threadsPerBlock,
         const char *dict,
         const int max_length,
         const int dict_size,
         const char *to_crack,
         int *correct_idx)
{
    // Call the kernel with the appropriate parameters.
    cudaCrackHashKernel<<<blocks, threadsPerBlock>>>(dict, max_length, dict_size, to_crack, correct_idx);
}
