#include "hip/hip_runtime.h"
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hashiru_cuda.cuh"

// TODO: Write kernels
__global__ void cudaCrackHashKernel(char *dict, const int max_length, const int dict_size, const char *to_crack, int *correct_idx)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char *current, *cur_hash;
    while(idx < dict_size)
    {
        current = dict + idx * max_length;
        //TODO: cur_hash = salsa20_gpu(current);
        //TODO: if(strcmp(cur_hash, to_crack) == 0)
        if(0)
        {
            *correct_idx = idx;
            break;
        }
        idx += blockDim.x * gridDim.x;
    }
}

void cudaCallCrackHashKernel(const unsigned int blocks,
         const unsigned int threadsPerBlock,
         char *dict,
         const int max_length,
         const int dict_size,
         const char *to_crack,
         int *correct_idx)
{
    // TODO:  cudaCrackHashKernel<<<blocks, threadsPerBlock>>>(dict, max_length, dict_size, to_crack, correct_idx);
}
