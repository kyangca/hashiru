#include "hip/hip_runtime.h"
#include <cstdio>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hashiru_cuda.cuh"

__global__ void cudaCrackHashKernel(const char *dict, const int max_length, const int dict_size, const char *to_crack, int *correct_idx)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char *current, *cur_hash, *c1, *c2;
    int equal;
    while(idx < dict_size)
    {
        current = (char *) (dict + idx * max_length);
        //TODO: cur_hash = salsa20_gpu(current);
        //TODO: if(strcmp(cur_hash, to_crack) == 0)
        c1 = cur_hash;
        c2 = (char *)to_crack;
        equal = 0;
        while(*c1 != '\0' && *c2 != '\0')
        {
            if(*c1 != *c2)
            {
                equal = 1;
                break;
            }
            c1++;
            c2++;
        }
        equal = ((*c1 != '\0') || (*c2 != '\0'));
        if(equal == 0)
        {
            *correct_idx = idx;
            break;
        }
        idx += blockDim.x * gridDim.x;
    }
}

void cudaCallCrackHashKernel(const unsigned int blocks,
         const unsigned int threadsPerBlock,
         const char *dict,
         const int max_length,
         const int dict_size,
         const char *to_crack,
         int *correct_idx)
{
    // TODO:  cudaCrackHashKernel<<<blocks, threadsPerBlock>>>(dict, max_length, dict_size, to_crack, correct_idx);
}
